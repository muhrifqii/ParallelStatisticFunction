
#include <hip/hip_runtime.h>

// ParallelSPSS.Form1
extern "C" __global__  void addVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);

// ParallelSPSS.Form1
extern "C" __global__  void addVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
